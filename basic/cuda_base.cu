/* coding: utf-8 */
/*
Author: Lin-Xing Zeng
Email:  jasonphysics@outlook.com | jasonphysics19@gmail.com

This file is part of Math Addition, in ./basic/cuda_base.cu
*/
extern "C"{

#include<wchar.h>
#include<stdint.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include"basic.h"
#include"cuda_base.cuh"

int Madd_N_cuda_GPU(void)
{
    int count;
    hipError_t res = hipGetDeviceCount(&count);
    if (res != hipSuccess){
        wchar_t error_info[MADD_ERROR_INFO_LEN];
        const char *cuda_info = hipGetErrorString(res);
        swprintf(error_info, MADD_ERROR_INFO_LEN-1, L"Madd_N_cuda_GPU: cuda func hipGetDeviceCount reports an error: %hs.", cuda_info);
        Madd_Error_Add(MADD_ERROR, error_info);
    }
    return count;
}

Madd_cuda_Device_Properties Madd_cuda_Get_Device_Property(void)
{
    hipError_t res_count, res_property;
    Madd_cuda_Device_Properties dp;
    dp.devices = NULL;
    res_count = hipGetDeviceCount(&dp.n_device);
    if (res_count != hipSuccess){
        wchar_t error_info[MADD_ERROR_INFO_LEN];
        const char *cuda_info = hipGetErrorString(res_count);
        swprintf(error_info, MADD_ERROR_INFO_LEN-1, L"Madd_Get_cuda_Device_Property: cuda func hipGetDeviceCount reports an error: %hs.", cuda_info);
        Madd_Error_Add(MADD_ERROR, error_info);
    }
    dp.devices = (struct hipDeviceProp_t*)malloc(dp.n_device*sizeof(struct hipDeviceProp_t));

    int i_dev;
    for (i_dev=0; i_dev<dp.n_device; i_dev++){
        res_property = hipGetDeviceProperties(dp.devices+i_dev, i_dev);
        if (res_property != hipSuccess){
            wchar_t error_info[MADD_ERROR_INFO_LEN];
            swprintf(error_info, MADD_ERROR_INFO_LEN-1, L"Madd_Get_cuda_Device_Property: cuda func hipGetDeviceProperties reports an error: %hs.", hipGetErrorString(res_property));
            Madd_Error_Add(MADD_ERROR, error_info);
        }
    }
    return dp;
}

void Madd_cuda_Get_Device_Mem(int i_dev, size_t *free_mem, size_t *total_mem)
{
    int i_current_dev;
    hipGetDevice(&i_current_dev);
    hipSetDevice(i_dev);
    hipError_t res = hipMemGetInfo(free_mem, total_mem);
    if (res != hipSuccess){
        wchar_t error_info[MADD_ERROR_INFO_LEN];
        swprintf(error_info, MADD_ERROR_INFO_LEN-1, L"Madd_cuda_Device_Mem: cuda func hipMemGetInfo reports an error: %hs.", hipGetErrorString(res));
        Madd_Error_Add(MADD_ERROR, error_info);
    }
    hipSetDevice(i_current_dev);
}

void Madd_cuda_Device_Property_Destroy(Madd_cuda_Device_Properties dp)
{
    free(dp.devices);
}

void Madd_cudaMalloc_error(int ret, const char *func_name, size_t size_alloc, const char *var_name)
{
    wchar_t error_info[MADD_ERROR_INFO_LEN];
    switch (ret){
        case hipErrorInvalidValue:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: hipMalloc (hipErrorInvalidValue) unable to allocate %llu bytes on CUDA GPU for variable '%hs': one or more of the parameters passed to the API call is not within an acceptable range of values.", func_name, size_alloc, var_name);
            break;
        case hipErrorOutOfMemory:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: hipMalloc (hipErrorOutOfMemory) unable to allocate %llu bytes on CUDA GPU for variable '%hs': The API call failed because it was unable to allocate enough memory or other resources to perform the requested operation.", func_name, size_alloc, var_name);
            break;
        default:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: hipMalloc - unable to allocate %llu bytes on CUDA GPU for variable '%hs': returns an error 0x%x that Madd doesn't know.", func_name, size_alloc, var_name, ret);
    }
    Madd_Error_Add(MADD_ERROR, error_info);
}

void Madd_cudaSetStream_error(int ret, const char *func_name)
{
    wchar_t error_info[MADD_ERROR_INFO_LEN];
    switch (ret){
        case hipErrorInvalidValue:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: cudaSetStream (hipErrorInvalidValue): one or more of the parameters passed to the API call is not within an acceptable range of values.", func_name);
            break;
        default:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: cudaSetStream returns an error 0x%x that Madd doesn't know.", func_name, ret);
    }
    Madd_Error_Add(MADD_ERROR, error_info);
}

}