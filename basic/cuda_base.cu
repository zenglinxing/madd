/* coding: utf-8 */
/*
Author: Lin-Xing Zeng
Email:  jasonphysics@outlook.com | jasonphysics19@gmail.com

This file is part of Math Addition, in ./basic/cuda_base.cu
*/
extern "C"{

#include<wchar.h>
#include<stdint.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include"basic.h"
#include"cuda_base.cuh"

int Madd_N_cuda_GPU(void)
{
    int count;
    hipError_t res = hipGetDeviceCount(&count);
    if (res != hipSuccess){
        wchar_t error_info[MADD_ERROR_INFO_LEN];
        const char *cuda_info = hipGetErrorString(res);
        swprintf(error_info, MADD_ERROR_INFO_LEN-1, L"Madd_N_cuda_GPU: cuda func hipGetDeviceCount reports an error: %hs.", cuda_info);
        Madd_Error_Add(MADD_ERROR, error_info);
    }
    return count;
}

Madd_cuda_Device_Properties Madd_cuda_Get_Device_Property(void)
{
    hipError_t res_count, res_property;
    Madd_cuda_Device_Properties dp;
    dp.devices = NULL;
    res_count = hipGetDeviceCount(&dp.n_device);
    if (res_count != hipSuccess){
        wchar_t error_info[MADD_ERROR_INFO_LEN];
        const char *cuda_info = hipGetErrorString(res_count);
        swprintf(error_info, MADD_ERROR_INFO_LEN-1, L"Madd_Get_cuda_Device_Property: cuda func hipGetDeviceCount reports an error: %hs.", cuda_info);
        Madd_Error_Add(MADD_ERROR, error_info);
    }
    dp.devices = (struct hipDeviceProp_t*)malloc(dp.n_device*sizeof(struct hipDeviceProp_t));

    int i_dev;
    for (i_dev=0; i_dev<dp.n_device; i_dev++){
        res_property = hipGetDeviceProperties(dp.devices+i_dev, i_dev);
        if (res_property != hipSuccess){
            wchar_t error_info[MADD_ERROR_INFO_LEN];
            swprintf(error_info, MADD_ERROR_INFO_LEN-1, L"Madd_Get_cuda_Device_Property: cuda func hipGetDeviceProperties reports an error: %hs.", hipGetErrorString(res_property));
            Madd_Error_Add(MADD_ERROR, error_info);
        }
    }
    return dp;
}

void Madd_cuda_Get_Device_Mem(int i_dev, size_t *free_mem, size_t *total_mem)
{
    int i_current_dev;
    hipGetDevice(&i_current_dev);
    hipSetDevice(i_dev);
    hipError_t res = hipMemGetInfo(free_mem, total_mem);
    if (res != hipSuccess){
        wchar_t error_info[MADD_ERROR_INFO_LEN];
        swprintf(error_info, MADD_ERROR_INFO_LEN-1, L"Madd_cuda_Device_Mem: cuda func hipMemGetInfo reports an error: %s.", hipGetErrorString(res));
        Madd_Error_Add(MADD_ERROR, error_info);
    }
    hipSetDevice(i_current_dev);
}

void Madd_cuda_Device_Property_Destroy(Madd_cuda_Device_Properties dp)
{
    free(dp.devices);
}

}