#include "hip/hip_runtime.h"
/* coding: utf-8 */
/*
Author: Lin-Xing Zeng
Email:  jasonphysics@outlook.com | jasonphysics19@gmail.com

This file is part of Math Addition, in ./linalg/determinant.c
*/
#include<stdio.h>
#include<stdlib.h>
#include<stdint.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>
#include<hipsolver.h>
extern "C"{
    #include"linalg.h"
    #include"../basic/basic.h"
}

static inline void cuda_func_error(hipsolverStatus_t ret, const char *func_name, const char *func_cuda_name)
{
    if (ret == HIPSOLVER_STATUS_SUCCESS) return;
    wchar_t error_info[MADD_ERROR_INFO_LEN];
    switch (ret){
        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: %hs (HIPSOLVER_STATUS_NOT_INITIALIZED) The library was not initialized.", func_name, func_cuda_name);
            break;
        case HIPSOLVER_STATUS_INVALID_VALUE:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: %hs (HIPSOLVER_STATUS_INVALID_VALUE) Invalid parameters were passed (m,n<0 or lda<max(1,m)).", func_name, func_cuda_name);
            break;
        case HIPSOLVER_STATUS_ARCH_MISMATCH:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: %hs (HIPSOLVER_STATUS_ARCH_MISMATCH) The device only supports compute capability 5.0 and above.", func_name, func_cuda_name);
            break;
        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: %hs (HIPSOLVER_STATUS_NOT_INITIALIZED) An internal operation failed.", func_name, func_cuda_name);
            break;
        default:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: %hs returns an error 0x%x that Madd doesn't know.", func_name, func_cuda_name, ret);
    }
    Madd_Error_Add(MADD_ERROR, error_info);
}

#define DET_CUDA_GPU__ALGORITHM(num_type) \
{ \
    signed char sign = 1; \
    int n1 = n + 1, i; \
    num_type *p = matrix; \
    *res = 1; \
    for (i=0; i<n; i++, p+=n1){ \
        if (ipiv[i] != i+1) sign *= -1; \
        *res *= *p; \
        printf("cuda %d-th: %f\n", i, *p); \
    } \
    *res *= sign; \
} \

static __global__ void Determinant_GPU(int n, double *matrix, int *ipiv, double *res)
DET_CUDA_GPU__ALGORITHM(double)

static __global__ void Determinant_GPU_f32(int n, float *matrix, int *ipiv, float *res)
DET_CUDA_GPU__ALGORITHM(float)

#define DET_CNUM_CUDA_GPU__ALGORITHM(num_type, Cnum_Mul, Cnum_Mul_Real) \
{ \
    signed char sign = 1; \
    int n1 = n + 1, i; \
    num_type *p = matrix; \
    res->real = 1; \
    res->imag = 0; \
    for (i=0; i<n; i++, p+=n1){ \
        if (ipiv[i] != i+1) sign *= -1; \
        *res = Cnum_Mul(*res, *p); \
    } \
    *res = Cnum_Mul_Real(*res, sign); \
} \

static __global__ void Determinant_GPU_c64(int n, Cnum *matrix, int *ipiv, Cnum *res)
DET_CNUM_CUDA_GPU__ALGORITHM(Cnum, Cnum_Mul, Cnum_Mul_Real)

static __global__ void Determinant_GPU_c32(int n, Cnum32 *matrix, int *ipiv, Cnum32 *res)
DET_CNUM_CUDA_GPU__ALGORITHM(Cnum32, Cnum_Mul_c32, Cnum_Mul_Real_c32)

extern "C"{

#define DET_CUDA__ALGORITHM(num_type, cuda_num_type, \
                            hipsolverDnDgetrf_bufferSize, hipsolverDnDgetrf, \
                            Determinant_GPU, \
                            func_getrf_buffer_name, func_getrf_name) \
{ \
    if (res == NULL){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: res is NULL.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (matrix == NULL){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: matrix is NULL.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (n == 0){ \
        *res = 0; \
        return true; \
    } \
 \
    uint64_t nn = (uint64_t)n*n; \
    size_t size_nn = nn*sizeof(num_type), size_ipiv = (uint64_t)n*sizeof(int); \
    num_type *d_matrix, *d_res; \
    int *d_ipiv, *d_info, info; \
    hipError_t error_matrix = hipMalloc(&d_matrix, size_nn+size_ipiv+sizeof(int)+sizeof(num_type)); \
    if (error_matrix != hipSuccess){ \
        Madd_cudaMalloc_error(error_matrix, __func__, size_nn+size_ipiv+sizeof(int)+sizeof(num_type), "d_matrix & d_ipiv & d_info & d_res"); \
        return false; \
    } \
    d_res = (num_type*)(d_matrix+nn); \
    d_ipiv = (int*)(d_res + 1); \
    d_info = (int*)(d_ipiv + n); \
    hipMemcpy(d_matrix, matrix, size_nn, hipMemcpyHostToDevice); \
 \
    hipStream_t stream; \
    hipError_t ret_stream = hipStreamCreate(&stream); \
    if (ret_stream != hipSuccess){ \
        hipFree(d_matrix); \
        Madd_cudaSetStream_error(ret_stream, __func__); \
        return false; \
    } \
    hipsolverHandle_t handle; \
    hipsolverStatus_t status_create = hipsolverDnCreate(&handle); \
    if (status_create != HIPSOLVER_STATUS_SUCCESS){ \
        hipFree(d_matrix); \
        hipStreamDestroy(stream); \
        Madd_cusolverDnCreate_error(status_create, __func__); \
        return false; \
    } \
    hipsolverSetStream(handle, stream); \
 \
    int lwork; \
    hipsolverStatus_t ret_buffer = hipsolverDnDgetrf_bufferSize( \
        handle, n, n, \
        (num_type*)d_matrix, n, \
        &lwork \
    ); \
    hipStreamSynchronize(stream); \
    if (ret_buffer != HIPSOLVER_STATUS_SUCCESS){ \
        hipFree(d_matrix); \
        hipStreamDestroy(stream); \
        hipsolverDnDestroy(handle); \
        cuda_func_error(ret_buffer, __func__, func_getrf_buffer_name); \
    } \
    cuda_num_type *d_workspace; \
    size_t size_workspace = (uint64_t)lwork*sizeof(cuda_num_type); \
    hipError_t ret_workspace = hipMalloc(&d_workspace, size_workspace); \
    if (ret_workspace != hipSuccess){ \
        hipFree(d_matrix); \
        hipStreamDestroy(stream); \
        hipsolverDnDestroy(handle); \
        Madd_cudaMalloc_error(ret_workspace, __func__, size_workspace, "d_workspace"); \
        return false; \
    } \
 \
    hipsolverStatus_t ret_getrf = hipsolverDnDgetrf( \
        handle, n, n, \
        (cuda_num_type*)d_matrix, n, \
        d_workspace, \
        d_ipiv, \
        d_info \
    ); \
    hipStreamSynchronize(stream); \
    if (ret_getrf != HIPSOLVER_STATUS_SUCCESS){ \
        hipFree(d_matrix); \
        hipStreamDestroy(stream); \
        hipsolverDnDestroy(handle); \
        hipFree(d_workspace); \
        cuda_func_error(ret_getrf, __func__, func_getrf_name); \
        return false; \
    } \
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost); \
    if (info < 0){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: %hs: the %d-th parameter is wrong (not counting handle)", __func__, func_getrf_name, -info); \
        hipFree(d_matrix); \
        hipStreamDestroy(stream); \
        hipsolverDnDestroy(handle); \
        hipFree(d_workspace); \
        return false; \
    } \
    if (info > 0){ \
        /* this mean matrix[info, info] = 0 */ \
        *res = 0; \
        hipFree(d_matrix); \
        hipStreamDestroy(stream); \
        hipsolverDnDestroy(handle); \
        hipFree(d_workspace); \
        return true; \
    } \
 \
    Determinant_GPU<<<1, 1, 0, stream>>>(n, d_matrix, d_ipiv, d_res); \
    hipStreamSynchronize(stream); \
 \
    hipMemcpy(res, d_res, sizeof(num_type), hipMemcpyDeviceToHost); \
 \
    hipFree(d_matrix); \
    hipStreamDestroy(stream); \
    hipsolverDnDestroy(handle); \
    hipFree(d_workspace); \
    return true; \
} \

bool Determinant_cuda(int n, double *matrix, double *res)
DET_CUDA__ALGORITHM(double, double,
                    hipsolverDnDgetrf_bufferSize, hipsolverDnDgetrf,
                    Determinant_GPU,
                    "hipsolverDnDgetrf_bufferSize", "hipsolverDnDgetrf")

bool Determinant_cuda_f32(int n, float *matrix, float *res)
DET_CUDA__ALGORITHM(float, float,
                    hipsolverDnSgetrf_bufferSize, hipsolverDnSgetrf,
                    Determinant_GPU_f32,
                    "hipsolverDnSgetrf_bufferSize", "hipsolverDnSgetrf")

#define DET_CNUM_CUDA__ALGORITHM(num_type, cuda_num_type, \
                                 hipsolverDnZgetrf_bufferSize, hipsolverDnZgetrf, \
                                 Determinant_GPU, \
                                 func_getrf_buffer_name, func_getrf_name) \
{ \
    if (res == NULL){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: res is NULL.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (matrix == NULL){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: matrix is NULL.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (n == 0){ \
        res->real = res->imag = 0; \
        return true; \
    } \
 \
    uint64_t nn = (uint64_t)n*n; \
    size_t size_nn = nn*sizeof(num_type), size_ipiv = (uint64_t)n*sizeof(int); \
    num_type *d_matrix, *d_res; \
    int *d_ipiv, *d_info, info; \
    hipError_t error_matrix = hipMalloc(&d_matrix, size_nn+size_ipiv+sizeof(int)+sizeof(num_type)); \
    if (error_matrix != hipSuccess){ \
        Madd_cudaMalloc_error(error_matrix, __func__, size_nn+size_ipiv+sizeof(int)+sizeof(num_type), "d_matrix & d_ipiv & d_info & d_res"); \
        return false; \
    } \
    d_res = (num_type*)(d_matrix+nn); \
    d_ipiv = (int*)(d_res + 1); \
    d_info = (int*)(d_ipiv + n); \
    hipMemcpy(d_matrix, matrix, size_nn, hipMemcpyHostToDevice); \
 \
    hipStream_t stream; \
    hipError_t ret_stream = hipStreamCreate(&stream); \
    if (ret_stream != hipSuccess){ \
        hipFree(d_matrix); \
        Madd_cudaSetStream_error(ret_stream, __func__); \
        return false; \
    } \
    hipsolverHandle_t handle; \
    hipsolverStatus_t status_create = hipsolverDnCreate(&handle); \
    if (status_create != HIPSOLVER_STATUS_SUCCESS){ \
        hipFree(d_matrix); \
        hipStreamDestroy(stream); \
        Madd_cusolverDnCreate_error(status_create, __func__); \
        return false; \
    } \
    hipsolverSetStream(handle, stream); \
 \
    int lwork; \
    hipsolverStatus_t ret_buffer = hipsolverDnZgetrf_bufferSize( \
        handle, n, n, \
        (cuda_num_type*)d_matrix, n, \
        &lwork \
    ); \
    hipStreamSynchronize(stream); \
    if (ret_buffer != HIPSOLVER_STATUS_SUCCESS){ \
        hipFree(d_matrix); \
        hipStreamDestroy(stream); \
        hipsolverDnDestroy(handle); \
        cuda_func_error(ret_buffer, __func__, func_getrf_buffer_name); \
    } \
    cuda_num_type *d_workspace; \
    size_t size_workspace = (uint64_t)lwork*sizeof(cuda_num_type); \
    hipError_t ret_workspace = hipMalloc(&d_workspace, size_workspace); \
    if (ret_workspace != hipSuccess){ \
        hipFree(d_matrix); \
        hipStreamDestroy(stream); \
        hipsolverDnDestroy(handle); \
        Madd_cudaMalloc_error(ret_workspace, __func__, size_workspace, "d_workspace"); \
        return false; \
    } \
 \
    hipsolverStatus_t ret_getrf = hipsolverDnZgetrf( \
        handle, n, n, \
        (cuda_num_type*)d_matrix, n, \
        d_workspace, \
        d_ipiv, \
        d_info \
    ); \
    hipStreamSynchronize(stream); \
    if (ret_getrf != HIPSOLVER_STATUS_SUCCESS){ \
        hipFree(d_matrix); \
        hipStreamDestroy(stream); \
        hipsolverDnDestroy(handle); \
        hipFree(d_workspace); \
        cuda_func_error(ret_getrf, __func__, func_getrf_name); \
        return false; \
    } \
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost); \
    if (info < 0){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: %hs: the %d-th parameter is wrong (not counting handle)", __func__, func_getrf_name, -info); \
        hipFree(d_matrix); \
        hipStreamDestroy(stream); \
        hipsolverDnDestroy(handle); \
        hipFree(d_workspace); \
        return false; \
    } \
    if (info > 0){ \
        /* this mean matrix[info, info] = 0 */ \
        res->real = res->imag = 0; \
        hipFree(d_matrix); \
        hipStreamDestroy(stream); \
        hipsolverDnDestroy(handle); \
        hipFree(d_workspace); \
        return true; \
    } \
 \
    Determinant_GPU<<<1, 1, 0, stream>>>(n, d_matrix, d_ipiv, d_res); \
    hipStreamSynchronize(stream); \
 \
    hipMemcpy(res, d_res, sizeof(num_type), hipMemcpyDeviceToHost); \
 \
    hipFree(d_matrix); \
    hipStreamDestroy(stream); \
    hipsolverDnDestroy(handle); \
    hipFree(d_workspace); \
    return true; \
} \

bool Determinant_cuda_c64(int n, Cnum *matrix, Cnum *res)
DET_CNUM_CUDA__ALGORITHM(Cnum, hipDoubleComplex,
                         hipsolverDnZgetrf_bufferSize, hipsolverDnZgetrf,
                         Determinant_GPU_c64,
                         "hipsolverDnZgetrf_bufferSize", "hipsolverDnZgetrf")

bool Determinant_cuda_c32(int n, Cnum32 *matrix, Cnum32 *res)
DET_CNUM_CUDA__ALGORITHM(Cnum32, hipComplex,
                         hipsolverDnCgetrf_bufferSize, hipsolverDnCgetrf,
                         Determinant_GPU_c32,
                         "hipsolverDnCgetrf_bufferSize", "hipsolverDnCgetrf")

}