/* coding: utf-8 */
/*
Author: Lin-Xing Zeng
Email:  jasonphysics@outlook.com | jasonphysics19@gmail.com

This file is part of Math Addition, in ./linalg/matrix_multiply_64.cu
*/
#include<stdlib.h>
#include<stdint.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
extern "C"{
#include"linalg.h"
#include"../basic/basic.h"
}

#if __CUDACC_VER_MAJOR__ >=12

#define MATRIX_MULTIPLY_CUDA64__ALGORITHM(num_type, hipblasDgemm, Matrix_Transpose) \
{ \
    if (m == 0){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: m is 0.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (n == 0){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: n is 0.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (l == 0){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: l is 0.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (a == NULL){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: matrix a is NULL.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (b == NULL){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: matrix b is NULL.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (res == NULL){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: matrix res is NULL.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
 \
    /* memory */ \
    uint64_t mm = m, nn = n, ll = l; \
    size_t size_a = mm*ll*sizeof(num_type), size_b = ll*nn*sizeof(num_type), size_res = mm*nn*sizeof(num_type); \
    num_type *d_a, *d_b, *d_res; \
    hipError_t cuda_malloc_a, cuda_malloc_b, cuda_malloc_res; \
    cuda_malloc_a = hipMalloc(&d_a, size_a); \
    if (cuda_malloc_a == hipErrorOutOfMemory){ \
        return false; \
    } \
    cuda_malloc_b = hipMalloc(&d_b, size_b); \
    if (cuda_malloc_b == hipErrorOutOfMemory){ \
        hipFree(d_a); \
        return false; \
    } \
    cuda_malloc_res = hipMalloc(&d_res, size_res); \
    if (cuda_malloc_res == hipErrorOutOfMemory){ \
        hipFree(d_a); \
        hipFree(d_b); \
        return false; \
    } \
    hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice); \
    hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice); \
 \
    /* matrix multiply */ \
    hipStream_t stream; \
    hipblasHandle_t handle; \
    hipblasCreate(&handle); \
    hipStreamCreate(&stream); \
    hipblasSetStream(handle, stream); \
 \
    num_type alpha = 1, beta = 0; \
    hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, \
                m, n, l, \
                &alpha, \
                d_a, l, \
                d_b, n, \
                &beta, \
                d_res, m); \
    hipStreamSynchronize(stream); \
 \
    hipMemcpy(res, d_res, size_res, hipMemcpyDeviceToHost); \
    hipblasDestroy(handle); \
    hipFree(d_a); \
    hipFree(d_b); \
    hipFree(d_res); \
 \
    Matrix_Transpose(n, m, res); \
    return true; \
} \

#define MATRIX_MULTIPLY_CUDA64_CNUM__ALGORITHM(num_type, hipblasDgemm, Matrix_Transpose) \
{ \
    if (m == 0){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: m is 0.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (n == 0){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: n is 0.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (l == 0){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: l is 0.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (a == NULL){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: matrix a is NULL.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (b == NULL){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: matrix b is NULL.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (res == NULL){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: matrix res is NULL.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
 \
    /* memory */ \
    uint64_t mm = m, nn = n, ll = l; \
    size_t size_a = mm*ll*sizeof(num_type), size_b = ll*nn*sizeof(num_type), size_res = mm*nn*sizeof(num_type); \
    num_type *d_a, *d_b, *d_res; \
    hipError_t cuda_malloc_a, cuda_malloc_b, cuda_malloc_res; \
    cuda_malloc_a = hipMalloc(&d_a, size_a); \
    if (cuda_malloc_a == hipErrorOutOfMemory){ \
        return false; \
    } \
    cuda_malloc_b = hipMalloc(&d_b, size_b); \
    if (cuda_malloc_b == hipErrorOutOfMemory){ \
        hipFree(d_a); \
        return false; \
    } \
    cuda_malloc_res = hipMalloc(&d_res, size_res); \
    if (cuda_malloc_res == hipErrorOutOfMemory){ \
        hipFree(d_a); \
        hipFree(d_b); \
        return false; \
    } \
    hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice); \
    hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice); \
 \
    /* matrix multiply */ \
    hipStream_t stream; \
    hipblasHandle_t handle; \
    hipblasCreate(&handle); \
    hipStreamCreate(&stream); \
    hipblasSetStream(handle, stream); \
 \
    num_type alpha, beta; \
    alpha.x = 1; \
    alpha.y = beta.x = beta.y = 0; \
    hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, \
                m, n, l, \
                &alpha, \
                d_a, l, \
                d_b, n, \
                &beta, \
                d_res, m); \
    hipStreamSynchronize(stream); \
 \
    hipMemcpy(res, d_res, size_res, hipMemcpyDeviceToHost); \
    hipblasDestroy(handle); \
    hipFree(d_a); \
    hipFree(d_b); \
    hipFree(d_res); \
 \
    Matrix_Transpose(n, m, res); \
    return true; \
} \

extern "C"{

bool Matrix_Multiply_cuda64(int64_t m, int64_t n, int64_t l,
                          double *a, double *b, double *res)
MATRIX_MULTIPLY_CUDA64__ALGORITHM(double, hipblasDgemm_64, Matrix_Transpose)

bool Matrix_Multiply_cuda64_f32(int64_t m, int64_t n, int64_t l,
                              float *a, float *b, float *res)
MATRIX_MULTIPLY_CUDA64__ALGORITHM(float, hipblasSgemm_64, Matrix_Transpose_f32)

bool Matrix_Multiply_cuda64_c64(int64_t m, int64_t n, int64_t l,
                              Cnum *a, Cnum *b, Cnum *res)
MATRIX_MULTIPLY_CUDA64_CNUM__ALGORITHM(hipDoubleComplex, hipblasZgemm_64, Matrix_Transpose_c64)

bool Matrix_Multiply_cuda64_c32(int64_t m, int64_t n, int64_t l,
                              Cnum32 *a, Cnum32 *b, Cnum32 *res)
MATRIX_MULTIPLY_CUDA64_CNUM__ALGORITHM(hipComplex, hipblasCgemm_64, Matrix_Transpose_c32)

}

#endif /* __CUDACC_VER_MAJOR__ >= 12 */