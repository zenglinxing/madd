#include "hip/hip_runtime.h"
/* coding: utf-8 */
/*
Author: Lin-Xing Zeng
Email:  jasonphysics@outlook.com | jasonphysics19@gmail.com

This file is part of Math Addition, in ./linalg/linear_equations_64.cu
check
https://docs.nvidia.com/cuda/cusolver/index.html#cusolverdnxgetrf
https://docs.nvidia.com/cuda/cusolver/index.html#cusolverdnxgetrs
*/
#include<stdlib.h>
#include<stdint.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>
#include<hipsolver.h>
extern "C"{
    #include"../basic/basic.h"
    #include"linalg.h"
}

// cUDA version should be >= 11.1
#if __CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 1)

static inline void Madd_cusolverDnXgetrf_error(hipsolverStatus_t ret, const char *func_name)
{
    if (ret == HIPSOLVER_STATUS_SUCCESS) return;
    wchar_t error_info[MADD_ERROR_INFO_LEN];
    switch (ret){
        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: hipsolverDnXgetrf_bufferSize (HIPSOLVER_STATUS_NOT_INITIALIZED) The library was not initialized.", func_name);
            break;
        case HIPSOLVER_STATUS_INVALID_VALUE:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: hipsolverDnXgetrf_bufferSize (HIPSOLVER_STATUS_INVALID_VALUE) Invalid parameters were passed (m,n<0 or lda<max(1,m)).", func_name);
            break;
        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: hipsolverDnXgetrf_bufferSize (HIPSOLVER_STATUS_INTERNAL_ERROR) An internal operation failed.", func_name);
            break;
        default:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: hipsolverDnXgetrf_bufferSize returns an error 0x%x that Madd doesn't know.", func_name, ret);
    }
    Madd_Error_Add(MADD_ERROR, error_info);
}

static inline void Madd_cusolverDnXgetrs_error(hipsolverStatus_t ret, const char *func_name)
{
    if (ret == HIPSOLVER_STATUS_SUCCESS) return;
    wchar_t error_info[MADD_ERROR_INFO_LEN];
    switch (ret){
        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: hipsolverDnXgetrf_bufferSize (HIPSOLVER_STATUS_NOT_INITIALIZED) The library was not initialized.", func_name);
            break;
        case HIPSOLVER_STATUS_INVALID_VALUE:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: hipsolverDnXgetrf_bufferSize (HIPSOLVER_STATUS_INVALID_VALUE) Invalid parameters were passed (n<0 or lda<max(1,n) or ldb<max(1,n)).", func_name);
            break;
        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: hipsolverDnXgetrf_bufferSize (HIPSOLVER_STATUS_INTERNAL_ERROR) An internal operation failed.", func_name);
            break;
        default:
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: hipsolverDnXgetrf_bufferSize returns an error 0x%x that Madd doesn't know.", func_name, ret);
    }
    Madd_Error_Add(MADD_ERROR, error_info);
}

#define LINEAR_EQUATIONS_CUDA64__ALGORITHM(num_type, HIP_R_64F, Matrix_Transpose) \
{ \
    if (n == 0){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: n is 0.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (matrix == NULL){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: matrix is NULL.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (n_vector == 0){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: n_vector is 0.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (vector == NULL){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: vector is NULL.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
 \
    size_t size_nn = (uint64_t)n*n*sizeof(num_type), size_nvec = (uint64_t)n*n_vector*sizeof(num_type); \
    num_type *d_matrix, *d_vector; \
    hipError_t error_matrix = hipMalloc(&d_matrix, size_nn + size_nvec); \
    if (error_matrix != hipSuccess){ \
        Madd_cudaMalloc_error(error_matrix, __func__); \
        return false; \
    } \
    d_vector = d_matrix + (uint64_t)n * n; \
 \
    Matrix_Transpose(n, n, matrix); \
    hipMemcpy(d_matrix, matrix, size_nn, hipMemcpyHostToDevice); \
    Matrix_Transpose(n, n, matrix); \
    Matrix_Transpose(n, n_vector, vector); \
    hipMemcpy(d_vector, vector, size_nvec, hipMemcpyHostToDevice); \
    /*Matrix_Transpose(n_vector, n, vector);*/ \
 \
    hipStream_t stream; \
    hipError_t ret_stream = hipStreamCreate(&stream); \
    if (ret_stream != hipSuccess){ \
        hipFree(d_matrix); \
        Madd_cudaSetStream_error(ret_stream, __func__); \
        return false; \
    } \
    hipsolverHandle_t handle; \
    hipsolverStatus_t status_create = hipsolverDnCreate(&handle); \
    if (status_create != HIPSOLVER_STATUS_SUCCESS){ \
        hipFree(d_matrix); \
        Madd_cusolverDnCreate_error(status_create, __func__); \
        return false; \
    } \
    hipsolverSetStream(handle, stream); \
 \
    hipsolverDnParams_t params = NULL; \
    hipsolverStatus_t ret_create_params = hipsolverDnCreateParams(&params); \
    if (ret_create_params != HIPSOLVER_STATUS_SUCCESS){ \
        hipFree(d_matrix); \
        hipsolverDnDestroy(handle); \
        Madd_cusolverDnCreateParams_error(ret_create_params, __func__); \
        return false; \
    } \
    hipsolverStatus_t ret_set_params = hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0); \
    if (ret_set_params != HIPSOLVER_STATUS_SUCCESS){ \
        hipFree(d_matrix); \
        hipsolverDnDestroy(handle); \
        hipsolverDnDestroyParams(params); \
        Madd_cusolverDnSetAdvOptions_error(ret_set_params, __func__); \
        return false; \
    } \
 \
    /* get buffer sizes */ \
    size_t workspaceInBytesOnDevice, workspaceInBytesOnHost; \
    hipsolverStatus_t ret_trf_buffer = hipsolverDnXgetrf_bufferSize( \
        handle, params, n, n, HIP_R_64F, \
        d_matrix, n, HIP_R_64F, \
        &workspaceInBytesOnDevice, &workspaceInBytesOnHost \
    ); \
    hipStreamSynchronize(stream); \
    if (ret_trf_buffer != HIPSOLVER_STATUS_SUCCESS){ \
        hipFree(d_matrix); \
        hipsolverDnDestroy(handle); \
        hipsolverDnDestroyParams(params); \
        Madd_cusolverDnXgetrf_error(ret_trf_buffer, __func__); \
        return false; \
    } \
    void *bufferOnDevice = NULL, *bufferOnHost = NULL; \
    hipError_t error_dev_buffer = hipMalloc(&bufferOnDevice, workspaceInBytesOnDevice); \
    if (error_dev_buffer != hipSuccess){ \
        hipFree(d_matrix); \
        hipsolverDnDestroy(handle); \
        hipsolverDnDestroyParams(params); \
        Madd_cudaMalloc_error(error_dev_buffer, __func__); \
        return false; \
    } \
    bufferOnHost = malloc(workspaceInBytesOnHost); \
    if (workspaceInBytesOnHost && bufferOnHost){ \
        hipFree(d_matrix); \
        hipsolverDnDestroy(handle); \
        hipsolverDnDestroyParams(params); \
        hipFree(bufferOnDevice); \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: unable to malloc %llu bytes for bufferOnHost.", __func__, workspaceInBytesOnHost); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
 \
    /* LU */ \
    int64_t *d_ipiv; \
    int info, *d_info; \
    hipError_t error_ipiv = hipMalloc(&d_ipiv, (uint64_t)n*sizeof(int64_t) + sizeof(int)); \
    if (error_ipiv != hipSuccess){ \
        hipFree(d_matrix); \
        hipsolverDnDestroy(handle); \
        hipsolverDnDestroyParams(params); \
        hipFree(bufferOnDevice); \
        free(bufferOnHost); \
        Madd_cudaMalloc_error(error_ipiv, __func__); \
        return false; \
    } \
    d_info = (int*)(d_ipiv + n); \
    hipsolverStatus_t ret_getrf = hipsolverDnXgetrf( \
        handle, params, n, n, HIP_R_64F, \
        d_matrix, n, d_ipiv, HIP_R_64F, \
        bufferOnDevice, workspaceInBytesOnDevice, \
        bufferOnHost, workspaceInBytesOnHost, \
        d_info \
    ); \
    hipStreamSynchronize(stream); \
    if (ret_getrf != HIPSOLVER_STATUS_SUCCESS){ \
        hipFree(d_matrix); \
        hipsolverDnDestroy(handle); \
        hipsolverDnDestroyParams(params); \
        hipFree(bufferOnDevice); \
        free(bufferOnHost); \
        hipFree(d_ipiv); \
        Madd_cusolverDnXgetrf_error(ret_getrf, __func__); \
        return false; \
    } \
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost); \
 \
    /* linear equation */ \
    hipsolverStatus_t ret_getrs = hipsolverDnXgetrs( \
        handle, params, HIPBLAS_OP_N, n, n_vector, HIP_R_64F, \
        d_matrix, n, d_ipiv, HIP_R_64F, \
        d_vector, n, d_info \
    ); \
    hipStreamSynchronize(stream); \
    if (ret_getrs != HIPSOLVER_STATUS_SUCCESS){ \
        hipFree(d_matrix); \
        hipsolverDnDestroy(handle); \
        hipsolverDnDestroyParams(params); \
        hipFree(bufferOnDevice); \
        free(bufferOnHost); \
        hipFree(d_ipiv); \
        Madd_cusolverDnXgetrs_error(ret_getrs, __func__); \
        return false; \
    } \
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost); \
    if (info != 0){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        if (info < 0){ \
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: the %d-th parameter is wrong (not counting handle).", __func__, -info); \
        }else{ \
            swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: up to now, Madd developer never saw the NVIDIA doc explains why info (0x%x) from hipsolverDnXgetrs is greater than 0.", __func__, info); \
        } \
        Madd_Error_Add(MADD_ERROR, error_info); \
    } \
 \
    hipMemcpy(vector, d_vector, size_nvec, hipMemcpyDeviceToHost); \
    Matrix_Transpose(n_vector, n, vector); \
 \
    hipsolverDnDestroyParams(params); \
    hipFree(d_matrix); \
    hipFree(bufferOnDevice); \
    hipFree(d_ipiv); \
    free(bufferOnHost); \
    hipsolverDnDestroy(handle); \
    return true; \
} \

extern "C"{

bool Linear_Equations_cuda64(int64_t n, double *matrix, int64_t n_vector, double *vector)
LINEAR_EQUATIONS_CUDA64__ALGORITHM(double, HIP_R_64F, Matrix_Transpose)

bool Linear_Equations_cuda64_f32(int64_t n, float *matrix, int64_t n_vector, float *vector)
LINEAR_EQUATIONS_CUDA64__ALGORITHM(float, HIP_R_32F, Matrix_Transpose_f32)

bool Linear_Equations_cuda64_c64(int64_t n, Cnum *matrix, int64_t n_vector, Cnum *vector)
LINEAR_EQUATIONS_CUDA64__ALGORITHM(Cnum, HIP_C_64F, Matrix_Transpose_c64)

bool Linear_Equations_cuda64_c32(int64_t n, Cnum32 *matrix, int64_t n_vector, Cnum32 *vector)
LINEAR_EQUATIONS_CUDA64__ALGORITHM(Cnum32, HIP_C_32F, Matrix_Transpose_c32)

}

#endif /* __CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 1) */