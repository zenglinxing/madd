/* coding: utf-8 */
/*
Author: Lin-Xing Zeng
Email:  jasonphysics@outlook.com | jasonphysics19@gmail.com

This file is part of Math Addition, in ./fft/fft.cu
*/
#include<stdlib.h>
#include<stdint.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>
#include<hipfft/hipfft.h>
extern "C"{
#include"fft.h"
#include"fft.cuh"
#include"../basic/basic.h"
}

#define FFT_CUDA__ALGORITHM(Cnum, cufft_type, \
                            hipfftDoubleComplex, \
                            hipfftExecZ2Z, cufftExecZ2Z_name, \
                            Cnum_Div) \
{ \
    if (n == 0){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: n is 0.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (arr == NULL){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: arr is NULL.", __func__); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
    if (fft_direction != MADD_FFT_FORWARD && fft_direction != MADD_FFT_INVERSE){ \
        wchar_t error_info[MADD_ERROR_INFO_LEN]; \
        swprintf(error_info, MADD_ERROR_INFO_LEN, L"%hs: fft_direction should be either MADD_FFT_FORWARD or MADD_FFT_INVERSE. You set %d.", __func__, fft_direction); \
        Madd_Error_Add(MADD_ERROR, error_info); \
        return false; \
    } \
 \
    int cufft_direction = (fft_direction == MADD_FFT_FORWARD) ? HIPFFT_FORWARD : HIPFFT_BACKWARD; \
 \
    /* cuda plan */ \
    hipfftHandle handle; \
    hipfftResult ret_plan = hipfftPlan1d(&handle, n, cufft_type, 1); \
    if (ret_plan != HIPFFT_SUCCESS){ \
        Madd_cufftPlan1d_error(ret_plan, __func__); \
        return false; \
    } \
    hipStream_t stream; \
    hipError_t ret_stream_create = hipStreamCreate(&stream); \
    if (ret_stream_create != hipSuccess){ \
        Madd_cudaSetStream_error(ret_stream_create, __func__); \
        return false; \
    } \
    hipfftSetStream(handle, stream); \
 \
    /* copy data */ \
    size_t size_arr = n*sizeof(hipfftDoubleComplex); \
    hipfftDoubleComplex *d_arr; \
    hipError_t ret_malloc = hipMalloc(&d_arr, size_arr); \
    if (ret_malloc != hipSuccess){ \
        Madd_cudaMalloc_error(ret_malloc, __func__, size_arr, "d_arr"); \
        return false; \
    } \
    hipMemcpy(d_arr, arr, size_arr, hipMemcpyHostToDevice); \
 \
    /* execute fft */ \
    hipfftResult ret_exec = hipfftExecZ2Z(handle, d_arr, d_arr, cufft_direction); \
    if (ret_exec != HIPFFT_SUCCESS){ \
        Madd_cufftExec_error(ret_exec, __func__, "hipfftExecZ2Z"); \
        hipFree(d_arr); \
        return false; \
    } \
    hipStreamSynchronize(stream); \
    hipfftDestroy(handle); \
 \
    hipMemcpy(arr, d_arr, size_arr, hipMemcpyDeviceToHost); \
    hipFree(d_arr); \
 \
    if (fft_direction == MADD_FFT_INVERSE){ \
        Cnum div; \
        div.real = n; \
        div.imag = 0; \
        for (uint64_t i=0; i<n; i++){ \
            arr[i] = Cnum_Div(arr[i], div); \
        } \
    } \
 \
    return true; \
} \

extern "C"{

bool Fast_Fourier_Transform_cuda(int n, Cnum *arr, int fft_direction)
FFT_CUDA__ALGORITHM(Cnum, HIPFFT_Z2Z,
                    hipfftDoubleComplex,
                    hipfftExecZ2Z, "hipfftExecZ2Z",
                    Cnum_Div)

bool Fast_Fourier_Transform_cuda_c32(int n, Cnum32 *arr, int fft_direction)
FFT_CUDA__ALGORITHM(Cnum32, HIPFFT_C2C,
                    hipfftComplex,
                    hipfftExecC2C, "hipfftExecC2C",
                    Cnum_Div_c32)

}